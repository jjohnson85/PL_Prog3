#include "hip/hip_runtime.h"
#include<cstdio>
#include<iostream>
#include<cmath>
#include<string>
#include "cudaPrime.cuh"

#define WARPSIZE 32
#define MAXTHREADS 1024
#define MAXBLOCKS 65535

using namespace std;

//test for a numbers primality
__global__ void isPrimeCoarse( ull offset, ull start, ull end, int * result )
{
	ull tid = threadIdx.x + blockIdx.x * blockDim.x;
	ull idx = tid + offset;
	ull val = tid + start;

    int prime = 1;
    if(val <= end)
    {
	    if( val < 2 )
	    {
		    prime = 0;
	    }
        else
        {
	        for( int i = 2; i < val / 2; i++ )
	        {
		        if( val % i == 0 )
		        {
			        prime = 0;
		        }
	        }
	    }
	    
        result[idx] = prime;
	}
}

//test for a numbers primality
__global__ void isPrimeFine( ull val, ull offset, int * result )
{
    //+2 to ignore 0, and 1
    bool prime = true;
    ull tid = threadIdx.x + blockIdx.x * blockDim.x + offset;
    if(tid == 0) *result = 1;
    if(val < 1) prime = false;
    else if(tid > 1 && tid < val)
    {
        if(val % tid == 0)
        {
            prime = false;
        }
    }
    if(tid == 0)
        if(__any(!prime)) *result = 0;
}

//test for a numbers primality
__global__ void isPrimeHybrid( ull store_offset, ull val_offset, int * result )
{
    ull tid = threadIdx.x;
    ull val = blockIdx.x + val_offset;
    ull idx = blockIdx.x + store_offset;

    if(tid == 0)result[idx] = 1;
    for(ull i=tid; i<val; i+=blockDim.x)
    {
        if(val % i == 0) result[idx] = 0;
    }
}

__global__ void reduce( int * data, ull size, ull gapSize )
{
    ull tid = threadIdx.x + blockIdx.x * blockDim.x;
    ull idx = tid*gapSize;
    ull offset = 16*gapSize;
    
    if(idx < size)
    {
        //printf("Thread %lli writing to index %lli\n", tid, idx);
        while(offset >= gapSize)
        {
            //printf("Offset %lli : %lli -> %lli\n", offset, idx, idx+offset);
            if(idx+offset < size)
            {
                //if(data[idx+offset] > 10000)
                //printf("Thread %lli adding index %lli: %i\n", tid, idx+offset, data[idx+offset]);
                data[idx] += data[idx+offset];
            }
            offset >>= 1;
        }
    }
}

string getCudaDeviceProperties()
{
    char out[500];
    int deviceNum;
    hipGetDevice( &deviceNum );
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, deviceNum);
    
    sprintf(out, "%s, CUDA %i.%i, %lli Mbytes global memory, %i CUDA cores", properties.name, properties.major, properties.minor, properties.totalGlobalMem/1024/1024, properties.maxThreadsPerBlock);
    return out;
}

ull sumRange(int * data, ull size, int warps)
{   
    //cout << "\nStarting cuda range sum on " << size << " items" << endl;

    int result;
	ull threadsPer = WARPSIZE*warps;
	
    bool more=false;
    ull threadRange = 1;
    ull threadsNeeded = (ull)(ceil((double)size/threadRange)+0.2);
    ull warpsNeeded = (ull)(ceil((double)threadsNeeded/WARPSIZE)+0.2);
	ull blocksNeeded = (ull)(ceil((double)warpsNeeded/warps)+0.2);
	ull blocks;
	ull blocksDone = 0;
	hipError_t lastError;
	
    do
    {
        //We'll need to reduce down to running
        //In a single warp
        more = warpsNeeded > 1;
        //cout << "Iteration " << count++ << endl;
        
        while(blocksDone < blocksNeeded)
        {
            //cout << blocksDone << "/" << blocksNeeded << " blocks" << endl;
            //Check if everything can be summed in one go, or if 
            //Multiple kernel calls needed
            if(blocksNeeded < MAXBLOCKS) blocks = blocksNeeded;
            else
            {
                blocks = MAXBLOCKS;
            }
            
            //cout << "Executing " << blocks << " blocks with " << threadsPer << " each" << endl;
            //cout << "Start is data[" << blocksDone*threadsPer << "], Size is " << size-blocksDone*threadsPer << " range is " << threadRange << endl;
            reduce<<<blocks, threadsPer>>>(data+blocksDone*threadsPer, size-blocksDone*threadsPer, threadRange);
            blocksDone += blocks;
        }
        threadRange = WARPSIZE*threadRange;
        threadsNeeded = (ull)(ceil((double)size/threadRange)+0.2);
        warpsNeeded = (ull)(ceil((double)threadsNeeded/WARPSIZE)+0.2);
	    blocksNeeded = (ull)(ceil((double)warpsNeeded/warps)+0.2);
        blocksDone = 0;
        
        hipDeviceSynchronize();
        lastError = hipPeekAtLastError();
        if(lastError != hipSuccess)
        {
            cout << "sumRange::Error during kernel execution: " << hipGetErrorString(lastError) << endl;
            return 0;
        }
    }while(more);

    hipMemcpy(&result, data, sizeof(int), hipMemcpyDeviceToHost);
    return result;
}

int runCudaCoarse( ull start, ull end, unsigned int warps )
{
	ull range = end-start+1;
	ull size = (range)*sizeof(int);
	ull threadsPer = warps*WARPSIZE;
	if(threadsPer > MAXTHREADS) threadsPer = MAXTHREADS;
	
	ull totalBlocks = (ull)(ceil((double)range/threadsPer)+0.2);
	hipError_t lastError;
	
	int *d_result;
	if(hipMalloc( &d_result, size )==hipErrorOutOfMemory)
	{
	    cout << "Error allocating memory on cuda device" << endl;
	    return 0;
	}
	int *result = (int *)malloc( size );
	
    ull threadsThisTime;
    ull totalDone=0;
    ull blocks;
	while(totalDone < range)
	{
	    threadsThisTime = MAXBLOCKS*threadsPer;
	    if(start+threadsThisTime > end) threadsThisTime = end-start+1;
	    blocks = (ull)(ceil(threadsThisTime/(double)threadsPer)+0.2);

        //cout << "IsPrimeCoarse<<<" << blocks << ", " << threadsPer << ">>>(" << totalDone << ", " << start << ", " << end << ", " << d_result << ")" << endl;
		isPrimeCoarse<<< blocks , threadsPer >>>( totalDone, start, end, d_result ); 
		    
		start += threadsThisTime;
		totalDone += threadsThisTime;
	}

    hipDeviceSynchronize();
    lastError = hipPeekAtLastError();
    if(lastError != hipSuccess)
    {
        cout << "runCudaCoarse::Error during kernel execution: " << hipGetErrorString(lastError) << endl;
        hipFree( d_result );
	    free( result );
        return 0;
    }
    
    int count = sumRange(d_result, range, warps);

	hipFree( d_result );
	free( result );
	
	return count;
}

int runCudaFine( ull start, ull end, unsigned int warps )
{
    ull range = end-start+1;
    ull size = (range)*sizeof(int);
    ull threadsPer = warps*WARPSIZE;
    
    if(threadsPer > MAXTHREADS) threadsPer = MAXTHREADS;
	hipError_t lastError;
    	
    int* d_result;
	if(hipMalloc( &d_result, size )==hipErrorOutOfMemory)
	{
	    cout << "Error allocating memory on cuda device" << endl;
	    return 0;
	}
    int* result = (int *)malloc( size );
        
    ull blocks;
    //i - index of return location
    //j - value to test
    int* i = d_result;
    for(ull j=start; j<=end; i++, j++)
    {
        //k - Offset from 0 for thread indices
        for(ull k=0; k<j; k+= threadsPer*MAXBLOCKS)
        {
            blocks = (ull)(ceil((double)j/threadsPer)+0.2);
            if(blocks > MAXBLOCKS) blocks = MAXBLOCKS;
            
            //cout << "IsPrimeFine<<<" << blocks << ", " << threadsPer << ">>>(" << j << ", " << k << ", " << i << ")" << endl;
            isPrimeFine<<<blocks, threadsPer>>>( j, k, i );
        }
    }

    hipDeviceSynchronize();
    lastError = hipPeekAtLastError();
    if(lastError != hipSuccess)
    {
        cout << "runCudaFine::Error during kernel execution: " << hipGetErrorString(lastError) << endl;
        hipFree( d_result );
	    free( result );
        return 0;
    }
    
    int count = sumRange(d_result, range, warps);
    
    hipFree( d_result );
    free( result );
    
    return count;
}

int runCudaHybrid( ull start, ull end, unsigned int warps )
{
    ull range = end-start+1;
    ull size = (range)*sizeof(int);
    ull threadsPer = warps*WARPSIZE;
    
    if(threadsPer > MAXTHREADS) threadsPer = MAXTHREADS;
	hipError_t lastError;
	
    int* d_result;
	if(hipMalloc( &d_result, size )==hipErrorOutOfMemory)
	{
	    cout << "Error allocating memory on cuda device" << endl;
	    return 0;
	}
    int* result = (int *)malloc( size );
    
    ull blocks;
    for(ull i=start, j=0; i<end; i+= MAXBLOCKS, j+=MAXBLOCKS)
    {
        blocks = end-i+1;
        if(blocks > MAXBLOCKS) blocks = MAXBLOCKS;
        
        //cout << "IsPrimeHybrid<<<" << blocks << ", " << threadsPer << ">>>(" << j << ", " << i << ", " << d_result << ")" << endl;
        isPrimeHybrid<<<blocks, threadsPer>>>(j, i, d_result);
    }

    hipDeviceSynchronize();
    lastError = hipPeekAtLastError();
    if(lastError != hipSuccess)
    {
        cout << "runCudaHybrid::Error during kernel execution: " << hipGetErrorString(lastError) << endl;
        hipFree( d_result );
	    free( result );
        return 0;
    }
    int count = sumRange(d_result, range, warps);
    
    hipFree( d_result );
    free( result );
    
    return count;
}
