#include "hip/hip_runtime.h"

#include<iostream>
#include<cmath>

#include "cudaPrime.cuh"

using namespace std;

//test for a numbers primality
__global__ void isPrime( unsigned long long p, unsigned long long end, int * result )
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int test = tid + p;

    if(test < end)
    {
	    bool prime = true;

	    if( test < 2 )
	    {
		    prime = false;
		    result[tid] = false;
		    return;
	    }

	    for( int i = 2; i < test / 2; i++ )
	    {
		    if( test % i == 0 )
		    {
			    prime = false;
		    }
	    }

	    result[tid] = prime;
	}
}

int runCuda( unsigned long long start, unsigned long long end )
{
	unsigned int range = end-start;
	unsigned int lastRangeTested = 0;
	unsigned int size = (range)*sizeof(int);
	unsigned int numblocks = 60000;
	unsigned int numcalls = 1;
	unsigned int numthreads = 32;
	int count = 0;

	int *result = (int *)malloc( size );
	int *d_result;
	hipMalloc( (int**)&d_result, size );
	//65535 max blocks

	numblocks = range;
	while( numblocks > 60000 )
	{
		numblocks -= 60000;
		numcalls += 1;
	}

	numblocks = range;
	for( int i = 0; i < numcalls; i++ )
	{
		isPrime<<< (int)ceil(numblocks/(double)numthreads) , numthreads >>>( i, end, d_result ); 
		lastRangeTested += numblocks * 32;
		numblocks -= 60000;
	}

	hipMemcpy( result, d_result, size, hipMemcpyDeviceToHost );

	for( int i = 0; i < range; i++ )
	{
		if( result[i] == true )
		{
			count++;
		}
	}

	hipFree( d_result );
	free( result );
	return count;
}
